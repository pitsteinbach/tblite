#include <hip/hip_runtime.h>
#include "hipblas.h"
#include <math.h>

// this scales the element of the first vector by the element of the second vector 
__global__ static void dElementwiseScale(double* vecinout, const double* vecin,size_t ndim2)
{
  // get the global id (in vector)
  size_t id = blockIdx.x*blockDim.x+threadIdx.x; 
  if (id < ndim2) vecinout[id] *= vecin[id];
}

// this scales the element of the first vector by the element of the second vector 
__global__ static void sElementwiseScale(float* vecinout, const float* vecin,size_t ndim2)
{
  // get the global id (in vector)
  size_t id = blockIdx.x*blockDim.x+threadIdx.x; 
  if (id < ndim2) vecinout[id] *= vecin[id];
}
extern "C" void sHadamard(float* vecinout, const float* vecin, size_t ndim2){
    int blockSize, gridSize;
    blockSize = 512;
    // Number of thread blocks in grid
    gridSize = (int)ceil((float)ndim2/blockSize);

    sElementwiseScale<<<gridSize,blockSize>>>(vecinout, vecin, ndim2);
}

extern "C" void dHadamard(double* vecinout, const double* vecin, size_t ndim2){
    int blockSize, gridSize;
    blockSize = 512;
    // Number of thread blocks in grid
    gridSize = (int)ceil((float)ndim2/blockSize);

    dElementwiseScale<<<gridSize,blockSize>>>(vecinout, vecin, ndim2);
}